#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include "ped_model.h"


// __constant__ int d_w[5 * 5];

namespace Ped {

    __global__ void heatFades(int* d_heatmap) {
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        // heat fades
        d_heatmap[tid] = (int)round(d_heatmap[tid] * 0.80);
    }

    __global__ void countHeatmap(int* d_heatmap,
                                float* d_desiredXs,
                                float* d_desiredYs,
                                const int agentSize) {
        /*
            Count how many agents want to go to each location
        */
        int tid = threadIdx.x + blockIdx.x * blockDim.x;

        // boundary check
        if (tid < agentSize) {
            int x = (int)d_desiredXs[tid];
            int y = (int)d_desiredYs[tid];
            if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
                return;
            atomicAdd(&d_heatmap[y*SIZE + x], 40);
        }
    }

    __global__ void colorHeatmap(int* d_heatmap,
                                float* d_desiredXs,
                                float* d_desiredYs, 
                                const int agentSize) {

        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (tid < agentSize) {
            int x = (int)d_desiredXs[tid];
            int y = (int)d_desiredYs[tid];

            atomicMin(&d_heatmap[y*SIZE + x], 255);
        }
    }

    __global__ void scaleHeatmap(int* d_heatmap, int* d_scaled_heatmap) {
        /*
            Scale the data for visual representation
            Parallize: each thread scale one heatmap pixel
        */
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        int value = d_heatmap[tid];
        int y = tid / SIZE;
        int x = tid % SIZE;
        for (int cellY = 0; cellY < CELLSIZE; cellY++) {
            for (int cellX = 0; cellX < CELLSIZE; cellX++){
                int s_y = y * CELLSIZE + cellY;
                int s_x = x * CELLSIZE + cellX;
                d_scaled_heatmap[s_y*SCALED_SIZE  + s_x] = value;
            }
        }

    }

    __global__ void filterHeatmap(int* d_scaled_heatmap,
                                int* d_blurred_heatmap) {
        /*
            Apply gaussian blur filter
            Parallize: parallelize the outer 2 for-loops
                dim3 filter_bSize(32, 32);
                dim3 filter_blocks(SCALED_SIZE / filter_bSize.x, SCALED_SIZE / filter_bSize.y);
        */
        __shared__ int shared_shm[32][32];

        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        
        shared_shm[threadIdx.y][threadIdx.x] = d_scaled_heatmap[y * SCALED_SIZE + x];
        
        __syncthreads();

        // register w
        const int r_w[5][5] = {
            { 1, 4, 7, 4, 1 },
            { 4, 16, 26, 16, 4 },
            { 7, 26, 41, 26, 7 },
            { 4, 16, 26, 16, 4 },
            { 1, 4, 7, 4, 1 }
        };

        if(2 <= x && x < SCALED_SIZE - 2 && 2 <= y && y < SCALED_SIZE - 2) {
            int sum = 0 ;
            for (int k = -2; k < 3; k++) {
                for (int l = -2; l < 3; l++) {
                    int shm_y = threadIdx.y + k;
                    int shm_x = threadIdx.x + l;
                    int v;
                    if(0 <= shm_y && shm_y < 32 && 0 <= shm_x && shm_x < 32)
                        v = shared_shm[shm_y][shm_x];
                    else
                        v = d_scaled_heatmap[(y + k) * SCALED_SIZE + x + l];
                    // sum += d_w[(2 + k) * 5 + (2 + l)] * v;
                    sum += r_w[2 + k][2 + l] * v;
                }
            }
            int val = sum / 273;
            d_blurred_heatmap[y * SCALED_SIZE + x] = 0x00FF0000 | val << 24;
        }   
    }

    void Model::setupHeatmapCUDA() {
        int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
        int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
        int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

        heatmap = (int**)malloc(SIZE*sizeof(int*));

        scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
        blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

        for (int i = 0; i < SIZE; i++) {
            heatmap[i] = hm + SIZE*i;
        }

        for (int i = 0; i < SCALED_SIZE; i++) {
            scaled_heatmap[i] = shm + SCALED_SIZE*i;
            blurred_heatmap[i] = bhm + SCALED_SIZE*i;
        }

        // Allocate and Copy to Device
        hipMalloc(&d_heatmap, SIZE*SIZE*sizeof(int));
        hipMemcpy(d_heatmap, heatmap[0], SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&d_scaled_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int));
        hipMemset(d_scaled_heatmap, 0, SCALED_SIZE*SCALED_SIZE*sizeof(int));

        hipMalloc(&d_blurred_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int));
        hipMemset(d_blurred_heatmap, 0, SCALED_SIZE*SCALED_SIZE*sizeof(int));

        // const int w[5][5] = {
        //     { 1, 4, 7, 4, 1 },
        //     { 4, 16, 26, 16, 4 },
        //     { 7, 26, 41, 26, 7 },
        //     { 4, 16, 26, 16, 4 },
        //     { 1, 4, 7, 4, 1 }
        // };

        // hipMemcpyToSymbol(HIP_SYMBOL(d_w), w, 5 * 5 * sizeof(int));

        // Allocate the desired Xs Ys on device
        hipMalloc(&d_desiredXs, agents.size()*sizeof(float));
        hipMalloc(&d_desiredYs, agents.size()*sizeof(float));
        printf("[DEBUG] CUDA heatmap setup!\n");
    }

    void Model::updateHeatmapCUDA() {

        // create streams
        hipStream_t dXStream;
        hipStream_t dYStream;
        hipStream_t mainStream;
        hipStreamCreate(&dXStream);
        hipStreamCreate(&dYStream);
        hipStreamCreate(&mainStream);


        hipEvent_t start[3], stop[3];
        for(int i = 0; i != 3; ++i) {
            hipEventCreate(&start[i]);
            hipEventCreate(&stop[i]);
        }

        // heatmap fades
        hipEventRecord(start[0], mainStream);

        dim3 hm_bSize = SIZE;
        dim3 hm_blocks = SIZE;
        heatFades<<<hm_blocks, hm_bSize, 0, mainStream>>>(d_heatmap);
        
        // heatmap count
        countHeatmap<<<hm_blocks, hm_bSize, 0, mainStream>>>(d_heatmap, d_desiredXs,
                                                        d_desiredYs, agents.size());

        // Color Heatmap
        colorHeatmap<<<hm_blocks, hm_bSize, 0, mainStream>>>(d_heatmap, d_desiredXs, d_desiredYs, agents.size());
        

        hipEventRecord(stop[0], mainStream);
        // hipEventSynchronize(stop[0]);
        // hipEventElapsedTime(&hmCreationCUDA, start[0], stop[0]);

        // copy desiredXs and desiredYs to device
        hipMemcpyAsync(d_desiredXs, h_desiredXs,
                        agents.size() * sizeof(float), hipMemcpyHostToDevice,
                        dXStream);

        hipMemcpyAsync(d_desiredYs, h_desiredYs,
                        agents.size() * sizeof(float), hipMemcpyHostToDevice,
                        dYStream);

        hipDeviceSynchronize();


        // Scale Heatmap
        hipEventRecord(start[1], mainStream);

        scaleHeatmap<<<hm_blocks, hm_bSize, 0, mainStream>>>(d_heatmap, d_scaled_heatmap);

        hipEventRecord(stop[1], mainStream);
        // hipEventSynchronize(stop[1]);
        // hipEventElapsedTime(&hmScalingCUDA, start[1], stop[1]);

        // Filter Heatmap
        hipEventRecord(start[2], mainStream);

        dim3 filter_bSize(32, 32);
        dim3 filter_blocks(SCALED_SIZE / filter_bSize.x, SCALED_SIZE / filter_bSize.y);

        filterHeatmap<<<filter_blocks, filter_bSize, 0, mainStream>>>(d_scaled_heatmap, d_blurred_heatmap);
        // __filterHeatmap<<<1, SIZE, 0, stream1>>>(d_scaled_heatmap, d_blurred_heatmap);

        hipMemcpyAsync(blurred_heatmap[0], d_blurred_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int), hipMemcpyDeviceToHost, mainStream);

        hipEventRecord(stop[2], mainStream);
        // hipEventSynchronize(stop[2]);
        // hipEventElapsedTime(&hmFilterCUDA, start[2], stop[2]);

        hipDeviceSynchronize();
        hipEventElapsedTime(&hmCreationCUDA, start[0], stop[0]);
        hipEventElapsedTime(&hmScalingCUDA, start[1], stop[1]);
        hipEventElapsedTime(&hmFilterCUDA, start[2], stop[2]);


        // destroy streams
        hipStreamDestroy(dXStream);
        hipStreamDestroy(dYStream);
        hipStreamDestroy(mainStream);

        for(int i = 0; i != 3; ++i) {
            hipEventDestroy(start[i]);
            hipEventDestroy(stop[i]);
        }

        hmCreationCUDATotal += hmCreationCUDA;
        hmScalingCUDATotal += hmScalingCUDA;
        hmFilterCUDATotal += hmFilterCUDA;

        printf("[CUDA] Creation: %fms, Scaling: %fms, filter: %fms\n", hmCreationCUDA, hmScalingCUDA, hmFilterCUDA);
        
    }

    void Model::freeCUDAMem() {
        hipFree(d_heatmap);
        hipFree(d_scaled_heatmap);
        hipFree(d_blurred_heatmap);
        hipFree(d_desiredXs);
        hipFree(d_desiredYs);


        free(heatmap);
        free(scaled_heatmap);
        free(blurred_heatmap);
    }


}  // namespace Ped