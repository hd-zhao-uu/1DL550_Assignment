#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include "ped_model.h"

// constexpr int BLOCK_NUMBER = 4;
// constexpr int BLOCK_SIZE = 256;
constexpr int WEIGHTSUM = 273;

__constant__ int dW[5 * 5];

namespace Ped {

__global__ void initHeatmap(int* hm, int** heatmap) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    heatmap[tid] = hm + SIZE * tid;
    // printf("initHeatmap\n");
}

__global__ void initScaledHeatmap(int* shm, int** scaled_heatmap) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    scaled_heatmap[tid] = shm + SCALED_SIZE * tid;
    // printf("initScaledHeatmap\n");
}

__global__ void initBlurredHeatmap(int* bhm, int** blurred_heatmap) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    blurred_heatmap[tid] = bhm + SCALED_SIZE * tid;
    // printf("initBlurredHeatmap\n");
}

__global__ void heatFades(int** heatmap) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int k = 0; k < SIZE; k++) {
        // heat fades
        heatmap[k][tid] = (int)round(heatmap[k][tid] * 0.80);
    }
    // printf("heatFades\n");
}

__global__ void countHeatmap(int** heatmap,
                             float* desiredXs,
                             float* desiredYs,
                             const int agentSize) {
    /*
        Count how many agents want to go to each location
    */
    // printf("-S countHeatmap\n");
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // boundary check
    if (tid < agentSize) {
        // printf("-IFS countHeatmap\n");
        int x = (int)desiredXs[tid];
        int y = (int)desiredYs[tid];
        printf("x=%d, y=%d\n", x, y);
        if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
            return;
        atomicAdd(&heatmap[y][x], 40);
        printf("-IFE countHeatmap\n");
    }
      
   //  printf("-E countHeatmap\n");
}

__global__ void colorHeatmap(int** heatmap,
                             float* desiredXs,
                             float* desiredYs, 
                             const int agentSize) {
    printf("-s colorHeatmap\n");
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < agentSize) {
        int x = (int)desiredXs[tid];
        int y = (int)desiredYs[tid];

        atomicMin(&heatmap[y][x], 255);
    }
    printf("-E colorHeatmap\n");
}

__global__ void scaleHeatmap(int** heatmap, int** scaled_heatmap) {
    /*
        Scale the data for visual representation
    */
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int x = 0; x < SIZE; x++) {
        int value = heatmap[tid][x];
        for (int cellY = 0; cellY < CELLSIZE; cellY++) {
            for (int cellX = 0; cellX < CELLSIZE; cellX++) {
                scaled_heatmap[tid * CELLSIZE + cellY][x * CELLSIZE + cellX] =
                    value;
            }
        }
    }

    // printf("scaleHeatmap\n");
}

__global__ void filterHeatmap(int** scaled_heatmap,
                              int** blurred_heatmap,
                              const int w[5][5]) {
    /*
        Apply gaussian blur filter
    */
   printf("filterHeatmap starts\n");
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= 2 && tid < SCALED_SIZE - 2) {
        for (int j = 2; j < SCALED_SIZE - 2; j++) {
            int sum = 0;
            for (int k = -2; k < 3; k++) {
                for (int l = -2; l < 3; l++) {
                    sum += w[2 + k][2 + l] * scaled_heatmap[tid + k][j + l];
                }
            }
            int value = sum / WEIGHTSUM;
            blurred_heatmap[tid][j] = 0x00FF0000 | value << 24;
        }
    }
    printf("filterHeatmap\n");
}

void Model::setupHeatmapCUDA() {
    printf("setupHeatmapCUDA Start\n");
    int *hm, *shm, *bhm;

    hipMalloc(&hm, SIZE * SIZE * sizeof(int));
    hipMemset(hm, 0, SIZE * SIZE);

    hipMalloc(&shm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
    hipMalloc(&heatmap, SIZE * sizeof(int*));

    hipMalloc(&scaled_heatmap, SCALED_SIZE * sizeof(int*));

    // blurred_heatmap shouldn't be in device
    hipHostMalloc(&bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
    hipHostMalloc(&blurred_heatmap, SCALED_SIZE * sizeof(int*));

    hipMalloc(&desiredXs, agents.size() * sizeof(float));
    hipMalloc(&desiredYs, agents.size() * sizeof(float));

    initHeatmap<<<1, SIZE>>>(hm, heatmap);
    hipDeviceSynchronize();

    initScaledHeatmap<<<CELLSIZE, SIZE>>>(shm, scaled_heatmap);
    hipDeviceSynchronize();

    initBlurredHeatmap<<<CELLSIZE, SIZE>>>(bhm, blurred_heatmap);
    hipDeviceSynchronize();

    for (int i = 0; i < SCALED_SIZE; i++) {
        blurred_heatmap[i] = bhm + SCALED_SIZE * i;
    }

    const int w[5][5] = {{1, 4, 7, 4, 1},
                         {4, 16, 26, 16, 4},
                         {7, 26, 41, 26, 7},
                         {4, 16, 26, 16, 4},
                         {1, 4, 7, 4, 1}};
    hipMemcpyToSymbol(HIP_SYMBOL(dW), w, 5 * 5 * sizeof(int));


    printf("setupHeatmapCUDA Ends\n");
}

void Model::updateHeatmapCUDA() {
    printf("--- updateHeatmapCUDA starts\n");

    // init stream
    hipStream_t fadeStream;
    hipStream_t countStream;
    hipStream_t otherStream;

    hipEvent_t fadeFinish;
    hipEvent_t countFinish;

    hipStreamCreate(&fadeStream);
    hipStreamCreate(&countStream);

    hipStreamCreate(&otherStream);

    hipEventCreate(&fadeFinish);
    hipEventCreate(&countFinish);

    // heatmap fades
    heatFades<<<1, SIZE, 0, fadeStream>>>(heatmap);
    hipEventRecord(fadeFinish, fadeStream);

    hipMemcpyAsync(desiredXs, agentSOA->xs,
                    agents.size() * sizeof(float), hipMemcpyHostToDevice,
                    countStream);
    hipMemcpyAsync(desiredYs, agentSOA->ys,
                    agents.size() * sizeof(float), hipMemcpyHostToDevice,
                    countStream);
    
    hipStreamWaitEvent(countStream, fadeFinish);
    printf("--- updateHeatmapCUDA copy ends\n");

    countHeatmap<<<CELLSIZE, SIZE, 0, countStream>>>(heatmap, desiredXs,
                                                     desiredYs, agents.size());
    hipEventRecord(countFinish, countStream);

    hipStreamWaitEvent(otherStream, countFinish);

    printf("-- color starts\n");
    // Color Heatmap
    colorHeatmap<<<1, SIZE, 0, otherStream>>>(heatmap, desiredXs, desiredYs, agents.size());

    // Scale Heatmap
    scaleHeatmap<<<1, SIZE, 0, otherStream>>>(heatmap, scaled_heatmap);

    // Apply gaussian blur filter
    constexpr int w[5][5] = {{1, 4, 7, 4, 1},
                             {4, 16, 26, 16, 4},
                             {7, 26, 41, 26, 7},
                             {4, 16, 26, 16, 4},
                             {1, 4, 7, 4, 1}};

    filterHeatmap<<<1, SIZE, 0, otherStream>>>(scaled_heatmap, blurred_heatmap,
                                               w);

    hipStreamSynchronize(otherStream);

    // destory events and streams
    hipStreamDestroy(fadeStream);
    hipStreamDestroy(countStream);
    hipStreamDestroy(otherStream);

    hipEventDestroy(fadeFinish);
    hipEventDestroy(countFinish);

    hipStreamDestroy(fadeStream);
    hipStreamDestroy(fadeStream);
    hipStreamDestroy(fadeStream);
    printf("--- updateHeatmapCUDA ends\n");
}
}  // namespace Ped